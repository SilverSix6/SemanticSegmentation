#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "types.h"

// CUDA kernel to reset clusters
__global__ void resetClusters(Cluster *clusters, int num_clusters) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_clusters) {
        clusters[i].x = 0;
        clusters[i].y = 0;
        clusters[i].l = 0;
        clusters[i].a = 0;
        clusters[i].b = 0;
        clusters[i].n = 0;
    }
}

// CUDA kernel to update clusters
__global__ void updateClusters(Cluster *clusters, int *segmentation_matrix, unsigned char *image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pixel_index = y * width + x;
        int cluster_idx = segmentation_matrix[pixel_index];

        atomicAdd((float *)&clusters[cluster_idx].x, (float)x);
        atomicAdd((float *)&clusters[cluster_idx].y, (float)y);
        atomicAdd((float *)&clusters[cluster_idx].l, (float)image[3 * pixel_index]);
        atomicAdd((float *)&clusters[cluster_idx].a, (float)image[3 * pixel_index + 1]);
        atomicAdd((float *)&clusters[cluster_idx].b, (float)image[3 * pixel_index + 2]);
        atomicAdd(&clusters[cluster_idx].n, 1);
    }
}

// CUDA kernel to normalize cluster values
__global__ void normalizeClusters(Cluster *clusters, int num_clusters) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_clusters) {
        int n = clusters[i].n;
        if (n > 0) {
            clusters[i].x /= n;
            clusters[i].y /= n;
            clusters[i].l /= n;
            clusters[i].a /= n;
            clusters[i].b /= n;
        }
    }
}

// Host function to launch kernels
void compute_cluster_centers(Cluster *h_clusters, int *h_segmentation_matrix, unsigned char *h_image, int width, int height, int num_clusters, int m) {
    Cluster *d_clusters;
    int *d_segmentation_matrix;
    unsigned char *d_image;

    // Allocate device memory
    hipMalloc(&d_clusters, num_clusters * sizeof(Cluster));
    hipMalloc(&d_segmentation_matrix, width * height * sizeof(int));
    hipMalloc(&d_image, 3 * width * height * sizeof(unsigned char));

    // Copy data to device
    hipMemcpy(d_clusters, h_clusters, num_clusters * sizeof(Cluster), hipMemcpyHostToDevice);
    hipMemcpy(d_segmentation_matrix, h_segmentation_matrix, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_image, h_image, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Kernel launch parameters
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Reset clusters
    resetClusters<<<(num_clusters + 255) / 256, 256>>>(d_clusters, num_clusters);
    hipDeviceSynchronize();

    // Update clusters
    updateClusters<<<gridDim, blockDim>>>(d_clusters, d_segmentation_matrix, d_image, width, height);
    hipDeviceSynchronize();

    // Normalize clusters
    normalizeClusters<<<(num_clusters + 255) / 256, 256>>>(d_clusters, num_clusters);
    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(h_clusters, d_clusters, num_clusters * sizeof(Cluster), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_clusters);
    hipFree(d_segmentation_matrix);
    hipFree(d_image);
}
