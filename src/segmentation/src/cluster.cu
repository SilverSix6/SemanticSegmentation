#include "hip/hip_runtime.h"
#include "cluster.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to reset clusters
__global__ void resetClusters(Cluster *clusters, int num_clusters) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_clusters) {
        clusters[i].x = 0;
        clusters[i].y = 0;
        clusters[i].l = 0;
        clusters[i].a = 0;
        clusters[i].b = 0;
        clusters[i].n = 0;
    }
}

__global__ void updateClusters(Cluster *clusters, int num_clusters, int *segmented_matrix, unsigned char *image, int width, int height) {
    __shared__ extern Cluster localClusters[];
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pixel_index = y * width + x;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // Initialize shared memory
    if (tid < num_clusters) {
        localClusters[tid].x = 0;
        localClusters[tid].y = 0;
        localClusters[tid].l = 0;
        localClusters[tid].a = 0;
        localClusters[tid].b = 0;
        localClusters[tid].n = 0;
    }
    __syncthreads();

    // Add pixels space and color value to it's corresponding clsuter based on the segmented matrix label
    if (x < width && y < height) {
        int cluster_idx = segmented_matrix[pixel_index];

        if (cluster_idx >= 0 && cluster_idx < num_clusters) {
            atomicAdd(&localClusters[cluster_idx].x, (float)x);
            atomicAdd(&localClusters[cluster_idx].y, (float)y);
            atomicAdd(&localClusters[cluster_idx].l, (float)image[3 * pixel_index]);
            atomicAdd(&localClusters[cluster_idx].a, (float)image[3 * pixel_index + 1]);
            atomicAdd(&localClusters[cluster_idx].b, (float)image[3 * pixel_index + 2]);
            atomicAdd(&localClusters[cluster_idx].n, 1);
        }
    }

    __syncthreads();

    // Merge local clusters into global memory
    if (tid < num_clusters) {
        atomicAdd(&clusters[tid].x, localClusters[tid].x);
        atomicAdd(&clusters[tid].y, localClusters[tid].y);
        atomicAdd(&clusters[tid].l, localClusters[tid].l);
        atomicAdd(&clusters[tid].a, localClusters[tid].a);
        atomicAdd(&clusters[tid].b, localClusters[tid].b);
        atomicAdd(&clusters[tid].n, localClusters[tid].n);
    }
}

// CUDA kernel to normalize cluster values
__global__ void normalizeClusters(Cluster *clusters, int num_clusters) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_clusters) {
        int n = clusters[i].n;
        if (n > 0) {
            clusters[i].x /= n;
            clusters[i].y /= n;
            clusters[i].l /= n;
            clusters[i].a /= n;
            clusters[i].b /= n;
        }
    }
}

/**
 * Computes a cluster's average spatial and color information based on it's pixels. 
 * 
 * @param h_clusters: A pointer to an array of clusters
 * @param h_segmented_matrix: A pointer to each pixel's cluster id. 
 * @param h_image: A pointer to the input image. This data should be stored in a 1D array in row-major order. The image is assumed to be in LAB color space.
 * @param width: The width of the image (number of pixels)
 * @param height: The height of the image (number of pixels)
 * @param num_clusters: The number of clusters in the h_clusters array
 * @param m: The compactness factor. Used to balance spatial and color proximity. Higher values enforce spatial uniformity.
 */
void compute_cluster_centers_cuda(Cluster *d_clusters, int *d_segmented_matrix, unsigned char *d_image, int width, int height, int num_clusters, int m) {

    // Kernel launch parameters
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Reset clusters
    resetClusters<<<(num_clusters + 255) / 256, 256>>>(d_clusters, num_clusters);
    hipDeviceSynchronize();

    // Update clusters
    updateClusters<<<gridDim, blockDim, num_clusters * sizeof(Cluster)>>>(d_clusters, num_clusters, d_segmented_matrix, d_image, width, height);
    hipDeviceSynchronize();

    // Normalize clusters
    normalizeClusters<<<(num_clusters + 255) / 256, 256>>>(d_clusters, num_clusters);
    hipDeviceSynchronize();

}

__global__ void initialize_cluster_centers(unsigned char *image, int width, int height, Cluster *clusters, int num_clusters) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_clusters) return;

    int grid_spacing = (int)sqrtf((float)(height * width) / num_clusters);
    int grid_x = index % (width / grid_spacing);
    int grid_y = index / (width / grid_spacing);

    int x = grid_spacing / 2 + grid_x * grid_spacing;
    int y = grid_spacing / 2 + grid_y * grid_spacing;
    
    if (x >= width || y >= height) return;
    
    int pixel_index = (y * width + x) * 3; // LAB is stored as 3 bytes per pixel

    clusters[index].l = image[pixel_index];
    clusters[index].a = image[pixel_index + 1];
    clusters[index].b = image[pixel_index + 2];
    clusters[index].x = x;
    clusters[index].y = y;
}

/**
 * Initializes a grid of clusters over the width of the image. The clusters are spaced evenly based on the image's size and the number of clusters.
 * 
 * @param image: A pointer to the input image. This data should be stored in a 1D array in row-major order. The image is assumed to be in LAB color space.
 * @param width: The width of the image (number of pixels)
 * @param height: The height of the image (number of pixels)
 * @param clusters: A pointer to an array of Clusters. This shoud be preallocated memory the size of num_clusters.
 * @param num_clusters: The size of the clusters array. 
 */
void initialize_cluster_centers_cuda(unsigned char *d_image, int width, int height, Cluster *d_clusters, int num_clusters) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_clusters + threadsPerBlock - 1) / threadsPerBlock;
    initialize_cluster_centers<<<blocksPerGrid, threadsPerBlock>>>(d_image, width, height, d_clusters, num_clusters);
    hipDeviceSynchronize(); // Ensure the kernel completes before returning
}


/**
 * Copys the cluster data from one array to another array. 
 * 
 * @param clusters: A pointer to an array of clusters
 * @param prev_clusters: A pointer to where the current cluster will be copied to.
 * @param num_clusters: The size of clusters array. clusters and prev_clusters should be the same size.
 */
void copy_cluster_cuda(Cluster *d_clusters, Cluster *d_prev_clusters, int num_clusters) {
    hipMemcpy(d_prev_clusters, d_clusters, num_clusters * sizeof(Cluster), hipMemcpyDeviceToDevice);
}