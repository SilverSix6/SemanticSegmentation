#include "hip/hip_runtime.h"
#include "cluster.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to reset clusters
__global__ void resetClusters(Cluster *clusters, int num_clusters) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_clusters) {
        clusters[i].x = 0;
        clusters[i].y = 0;
        clusters[i].l = 0;
        clusters[i].a = 0;
        clusters[i].b = 0;
        clusters[i].n = 0;
    }
}

__global__ void updateClusters(Cluster *clusters, int num_clusters, int *segmented_matrix, unsigned char *image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pixel_index = y * width + x;

    if (x < width && y < height) {
        int cluster_idx = segmented_matrix[pixel_index];

        if (cluster_idx >= 0 && cluster_idx < num_clusters) {
            atomicAdd(&clusters[cluster_idx].x, (float)x);
            atomicAdd(&clusters[cluster_idx].y, (float)y);
            atomicAdd(&clusters[cluster_idx].l, (float)image[3 * pixel_index]);
            atomicAdd(&clusters[cluster_idx].a, (float)image[3 * pixel_index + 1]);
            atomicAdd(&clusters[cluster_idx].b, (float)image[3 * pixel_index + 2]);
            atomicAdd(&clusters[cluster_idx].n, 1);
        }
    }
}


// CUDA kernel to normalize cluster values
__global__ void normalizeClusters(Cluster *clusters, int num_clusters) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_clusters) {
        int n = clusters[i].n;
        if (n > 0) {
            clusters[i].x /= n;
            clusters[i].y /= n;
            clusters[i].l /= n;
            clusters[i].a /= n;
            clusters[i].b /= n;
        }
    }
}

/**
 * Computes a cluster's average spatial and color information based on it's pixels. 
 * 
 * @param h_clusters: A pointer to an array of clusters
 * @param h_segmented_matrix: A pointer to each pixel's cluster id. 
 * @param h_image: A pointer to the input image. This data should be stored in a 1D array in row-major order. The image is assumed to be in LAB color space.
 * @param width: The width of the image (number of pixels)
 * @param height: The height of the image (number of pixels)
 * @param num_clusters: The number of clusters in the h_clusters array
 * @param m: The compactness factor. Used to balance spatial and color proximity. Higher values enforce spatial uniformity.
 */
void compute_cluster_centers_cuda(Cluster *d_clusters, int *d_segmented_matrix, unsigned char *d_image, int width, int height, int num_clusters, int m) {

    // Kernel launch parameters
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Reset clusters
    resetClusters<<<(num_clusters + 255) / 256, 256>>>(d_clusters, num_clusters);
    hipDeviceSynchronize();

    // Update clusters
    updateClusters<<<gridDim, blockDim, num_clusters * sizeof(Cluster)>>>(d_clusters, num_clusters, d_segmented_matrix, d_image, width, height);
    hipDeviceSynchronize();

    // Normalize clusters
    normalizeClusters<<<(num_clusters + 255) / 256, 256>>>(d_clusters, num_clusters);
    hipDeviceSynchronize();

}

/**
 * Computes a cluster's average spatial and color information based on it's pixels. 
 * 
 * @param h_clusters: A pointer to an array of clusters
 * @param h_segmented_matrix: A pointer to each pixel's cluster id. 
 * @param h_image: A pointer to the input image. This data should be stored in a 1D array in row-major order. The image is assumed to be in LAB color space.
 * @param width: The width of the image (number of pixels)
 * @param height: The height of the image (number of pixels)
 * @param num_clusters: The number of clusters in the h_clusters array
 * @param m: The compactness factor. Used to balance spatial and color proximity. Higher values enforce spatial uniformity.
 */
void compute_cluster_centers_cpu(Cluster *clusters, int *segmented_matrix, unsigned char *image, int width, int height, int num_clusters, int m) {

    // Reset cluster
    for (int i = 0; i < num_clusters; i++) {
        clusters[i].x = 0;
        clusters[i].y = 0;
        clusters[i].l = 0;
        clusters[i].a = 0;
        clusters[i].b = 0;
        clusters[i].n = 0;
    }

    // Update clusters with pixel values
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            int pixel_index = y * width + x;
            int cluster_idx = segmented_matrix[pixel_index];

            clusters[cluster_idx].x += x;
            clusters[cluster_idx].y += y;
            clusters[cluster_idx].l += image[3 * pixel_index];
            clusters[cluster_idx].a += image[3 * pixel_index + 1];
            clusters[cluster_idx].b += image[3 * pixel_index + 2];
            clusters[cluster_idx].n++;
        }
    }

    // Normalize Clusters
    for (int i = 0; i < num_clusters; i++) {
        int n = clusters[i].n;
        if (n > 0) {
            clusters[i].x /= n;
            clusters[i].y /= n;
            clusters[i].l /= n;
            clusters[i].a /= n;
            clusters[i].b /= n;
        }
    }

}

__global__ void initialize_cluster_centers(unsigned char *image, int width, int height, Cluster *clusters, int num_clusters) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_clusters) return;

    int grid_spacing = (int)sqrtf((float)(height * width) / num_clusters);
    int grid_x = index % (width / grid_spacing);
    int grid_y = index / (width / grid_spacing);

    int x = grid_spacing / 2 + grid_x * grid_spacing;
    int y = grid_spacing / 2 + grid_y * grid_spacing;
    
    if (x >= width || y >= height) return;
    
    int pixel_index = (y * width + x) * 3; // LAB is stored as 3 bytes per pixel

    clusters[index].l = image[pixel_index];
    clusters[index].a = image[pixel_index + 1];
    clusters[index].b = image[pixel_index + 2];
    clusters[index].x = x;
    clusters[index].y = y;
}

/**
 * Initializes a grid of clusters over the width of the image. The clusters are spaced evenly based on the image's size and the number of clusters.
 * 
 * @param image: A pointer to the input image. This data should be stored in a 1D array in row-major order. The image is assumed to be in LAB color space.
 * @param width: The width of the image (number of pixels)
 * @param height: The height of the image (number of pixels)
 * @param clusters: A pointer to an array of Clusters. This shoud be preallocated memory the size of num_clusters.
 * @param num_clusters: The size of the clusters array. 
 */
void initialize_cluster_centers_cuda(unsigned char *d_image, int width, int height, Cluster *d_clusters, int num_clusters) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_clusters + threadsPerBlock - 1) / threadsPerBlock;
    initialize_cluster_centers<<<blocksPerGrid, threadsPerBlock>>>(d_image, width, height, d_clusters, num_clusters);
    hipDeviceSynchronize(); // Ensure the kernel completes before returning
}

/**
 * Initializes a grid of clusters over the width of the image. The clusters are spaced evenly based on the image's size and the number of clusters.
 * 
 * @param image: A pointer to the input image. This data should be stored in a 1D array in row-major order. The image is assumed to be in LAB color space.
 * @param width: The width of the image (number of pixels)
 * @param height: The height of the image (number of pixels)
 * @param clusters: A pointer to an array of Clusters. This shoud be preallocated memory the size of num_clusters.
 * @param num_clusters: The size of the clusters array. 
 */
void initialize_cluster_centers_cpu(unsigned char *image, int width, int height, Cluster *clusters, int num_clusters) {
    if (!image || !clusters) return; // Prevent null pointer access

    int grid_spacing = (int)sqrtf((float)(height * width) / num_clusters);
    if (grid_spacing == 0) grid_spacing = 1; // Prevent division by zero

    int grid_size_x = width / grid_spacing;
    if (grid_size_x == 0) grid_size_x = 1; // Avoid division by zero

    for (int index = 0; index < num_clusters; index++) {
        int grid_x = index % grid_size_x;
        int grid_y = index / grid_size_x;

        int x = grid_spacing / 2 + grid_x * grid_spacing;
        int y = grid_spacing / 2 + grid_y * grid_spacing;
        
        if (x >= width || y >= height) continue;
        
        int pixel_index = (y * width + x) * 3; // LAB is stored as 3 bytes per pixel
    
        clusters[index].l = image[pixel_index];
        clusters[index].a = image[pixel_index + 1];
        clusters[index].b = image[pixel_index + 2];
        clusters[index].x = x;
        clusters[index].y = y;
    }
}

/**
 * Copys the cluster data from one array to another array. 
 * 
 * @param clusters: A pointer to an array of clusters
 * @param prev_clusters: A pointer to where the current cluster will be copied to.
 * @param num_clusters: The size of clusters array. clusters and prev_clusters should be the same size.
 */
void copy_cluster_cuda(Cluster *d_clusters, Cluster *d_prev_clusters, int num_clusters) {
    hipMemcpy(d_prev_clusters, d_clusters, num_clusters * sizeof(Cluster), hipMemcpyDeviceToDevice);
}

/**
 * Copys the cluster data from one array to another array. 
 * 
 * @param clusters: A pointer to an array of clusters
 * @param prev_clusters: A pointer to where the current cluster will be copied to.
 * @param num_clusters: The size of clusters array. clusters and prev_clusters should be the same size.
 */
void copy_cluster_cpu(Cluster *clusters, Cluster *prev_clusters, int num_clusters) {
    memcpy(prev_clusters, clusters, num_clusters * sizeof(Cluster));
}